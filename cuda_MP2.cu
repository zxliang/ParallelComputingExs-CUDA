#include "hip/hip_runtime.h"
#include "cuda_MP2.cuh"
#include "cuda_MP1.cuh"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
  //Multiply the two matrices
  unsigned row = threadIdx.x + blockDim.x * blockIdx.x;
  unsigned col = threadIdx.y + blockDim.y * blockIdx.y;

  if (row < MATRIX_SIZE && col < MATRIX_SIZE) {
	float sum = 0.0;
	for (int i = 0; i < MATRIX_SIZE; i++) {
	  sum += M.elements[row*MATRIX_SIZE+i] * N.elements[i*MATRIX_SIZE+col];
	}
	P.elements[row*MATRIX_SIZE + col] = sum;
  }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int cuda_MP2(int argc, char** argv) {

  // Matrices for the program
  Matrix  M;
  Matrix  N;
  Matrix  P;
  // Number of elements in the solution matrix
  // Assuming square matrices, so the sizes of M, N and P are equal
  unsigned int size_elements = WP * HP;
  int errorM = 0, errorN = 0;

  srand(2012);

  // Check command line for input matrix files
  if (argc != 3 && argc != 4)
  {
	// No inputs provided
	// Allocate and initialize the matrices
	M = AllocateMatrix(MATRIX_SIZE, MATRIX_SIZE, 1);
	N = AllocateMatrix(MATRIX_SIZE, MATRIX_SIZE, 1);
	P = AllocateMatrix(MATRIX_SIZE, MATRIX_SIZE, 0);
  }
  else
  {
	// Inputs provided
	// Allocate and read source matrices from disk
	M = AllocateMatrix(MATRIX_SIZE, MATRIX_SIZE, 0);
	N = AllocateMatrix(MATRIX_SIZE, MATRIX_SIZE, 0);
	P = AllocateMatrix(MATRIX_SIZE, MATRIX_SIZE, 0);
	errorM = ReadFile(&M, argv[1]);
	errorN = ReadFile(&N, argv[2]);
	// check for read errors
	if (errorM != size_elements || errorN != size_elements)
	{
	  printf("Error reading input files %d, %d\n", errorM, errorN);
	  return 1;
	}
  }

  // M * N on the device
  MatrixMulOnDevice(M, N, P);

  // compute the matrix multiplication on the CPU for comparison
  Matrix reference = AllocateMatrix(MATRIX_SIZE, MATRIX_SIZE, 0);
  computeGold(reference.elements, M.elements, N.elements, HM, WM, WN);

  // check if the device result is equivalent to the expected solution
  bool res = compareGold(reference.elements, P.elements, size_elements, 0.0001f);
  printf("Test %s\n", (true == res) ? "PASSED" : "FAILED");

  // output result if output file is requested
  if (argc == 4)
  {
	WriteFile(P, argv[3]);
  }
  else if (argc == 2)
  {
	WriteFile(P, argv[1]);
  }

  // Free host matrices
  free(M.elements);
  M.elements = NULL;
  free(N.elements);
  N.elements = NULL;
  free(P.elements);
  P.elements = NULL;

  return 0;
}

// Allocate a matrix of dimensions height*width
//	If init == 0, initialize to all zeroes.  
//	If init == 1, perform random initialization.
Matrix AllocateMatrix(int height, int width, int init)
{
  Matrix M;
  M.width = M.pitch = width;
  M.height = height;
  int size = M.width * M.height;
  M.elements = NULL;

  M.elements = (float*)malloc(size * sizeof(float));

  for (unsigned int i = 0; i < M.height * M.width; i++)
  {
	M.elements[i] = (init == 0) ? (0.0f) : (rand() / (float)RAND_MAX);
  }
  return M;
}

// Read a 16x16 floating point matrix in from file
int ReadFile(Matrix* M, char* file_name)
{
  unsigned int data_read = MATRIX_SIZE*MATRIX_SIZE;
//  cutReadFilef(file_name, &(M->elements), &data_read, true);
  ifstream iFile(file_name);
  unsigned i = 0;
  if (iFile) {
	float data;
	while (iFile >> data) {
	  M->elements[i++] = data;
	}
  }
  data_read = i;
  return data_read;
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void MatrixMulOnDevice(const Matrix M, const Matrix N, Matrix P)
{
  //Interface host call to the device kernel code and invoke the kernel

  Matrix dM_M = AllocateDeviceMatrix(M);
  Matrix dM_N = AllocateDeviceMatrix(N);
  Matrix dM_P = AllocateDeviceMatrix(P);

  CopyToDeviceMatrix(dM_M, M);
  CopyToDeviceMatrix(dM_N, N);

  dim3 dimGrid, dimBlock;

  dimGrid.x = dimGrid.y = dimGrid.z = 1;
  dimBlock.x = dimBlock.y = MATRIX_SIZE;
  dimBlock.z = 1;

  MatrixMulKernel<<<dimGrid, dimBlock>>>(dM_M, dM_N, dM_P);

  CopyFromDeviceMatrix(P, dM_P);

  hipFree(&dM_M);
  hipFree(&dM_N);
  hipFree(&dM_P);

}

// Allocate a device matrix of same size as M.
Matrix AllocateDeviceMatrix(const Matrix M)
{
  Matrix Mdevice = M;
  int size = M.width * M.height * sizeof(float);
  hipMalloc((void**)&Mdevice.elements, size);
  return Mdevice;
}

// Copy a host matrix to a device matrix.
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost)
{
  int size = Mhost.width * Mhost.height * sizeof(float);
  Mdevice.height = Mhost.height;
  Mdevice.width = Mhost.width;
  Mdevice.pitch = Mhost.pitch;
  hipMemcpy(Mdevice.elements, Mhost.elements, size,
	hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice)
{
  int size = Mdevice.width * Mdevice.height * sizeof(float);
  hipMemcpy(Mhost.elements, Mdevice.elements, size,
	hipMemcpyDeviceToHost);
}

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set
//! C = A * B
//! @param C          reference data, computed but preallocated
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param hA         height of matrix A
//! @param wB         width of matrix B
////////////////////////////////////////////////////////////////////////////////
void computeGold(float* C, const float* A, const float* B, unsigned int hA, 
  unsigned int wA, unsigned int wB)
{
  for (unsigned int i = 0; i < hA; ++i)
	for (unsigned int j = 0; j < wB; ++j) {
	  double sum = 0;
	  for (unsigned int k = 0; k < wA; ++k) {
		double a = A[i * wA + k];
		double b = B[k * wB + j];
		sum += a * b;
	  }
	  C[i * wB + j] = (float)sum;
	}
}

// Write a 16x16 floating point matrix to file
void WriteFile(Matrix M, char* file_name)
{
//  cutWriteFilef(file_name, M.elements, M.width*M.height, 0.0001f);
  ofstream oFile(file_name);
  if (oFile) {
	for (int i = 0; i < MATRIX_SIZE; i++) {
	  oFile << M.elements[i] << " ";
	}
	oFile.close();
  }
}

